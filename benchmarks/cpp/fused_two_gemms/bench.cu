#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "cutlass_fused_two_gemms.cuh"
#include "tilefusion_fused_two_gemms.cuh"
#include "util.cuh"

template <typename WholeShape, typename CtaTileShape, typename WarpLayout,
          const int kBatch, const int kSharedAccess>
void run(float epsilon = 1e-3) {
    using InType = __half;
    using AccType = float;

    static constexpr int kM = dim_size<0, WholeShape>;
    static constexpr int kN = dim_size<1, WholeShape>;
    static constexpr int kK = dim_size<2, WholeShape>;
    static constexpr int kP = dim_size<3, WholeShape>;

    static constexpr int kTM = dim_size<0, CtaTileShape>;
    static constexpr int kTN = dim_size<1, CtaTileShape>;
    static constexpr int kTK = dim_size<2, CtaTileShape>;
    static constexpr int kTP = dim_size<3, CtaTileShape>;

    static_assert(kK == kTK, "The current implementation requires kTK == K.");
    static_assert(kP == kTP, "The current implementation requires kTP == P.");

    static constexpr int kWarpPerRow = tl::num_rows<WarpLayout>;
    static constexpr int kWarpPerCol = tl::num_cols<WarpLayout>;

    thrust::host_vector<cutlass::half_t> h_a(kM * kK * kBatch);

    for (int i = 0; i < h_a.size(); ++i) {
        h_a[i] = static_cast<cutlass::half_t>(rand_float());
    }

    thrust::host_vector<cutlass::half_t> h_b(kK * kN * kBatch);
    for (int i = 0; i < h_b.size(); ++i) {
        h_b[i] = static_cast<cutlass::half_t>(rand_float());
    }

    thrust::host_vector<cutlass::half_t> h_c(kN * kP * kBatch);
    for (int i = 0; i < h_c.size(); ++i) {
        h_c[i] = static_cast<cutlass::half_t>(rand_float());
    }

    thrust::host_vector<InType> h_d(kM * kP * kBatch);
    thrust::fill(h_d.begin(), h_d.end(), 0.);

    thrust::host_vector<cutlass::half_t> h_d2(kM * kP * kBatch);
    thrust::fill(h_d2.begin(), h_d2.end(), 0.);

    thrust::host_vector<__half> h_d3(kM * kP * kBatch);
    thrust::fill(h_d3.begin(), h_d3.end(), 0.);

    thrust::device_vector<cutlass::half_t> d_a = h_a;
    thrust::device_vector<cutlass::half_t> d_b = h_b;
    thrust::device_vector<cutlass::half_t> d_c = h_c;
    thrust::device_vector<InType> d_d = h_d;
    thrust::device_vector<cutlass::half_t> d_d2 = h_d2;
    thrust::device_vector<__half> d_d3 = h_d3;

    const cutlass::half_t* CA = thrust::raw_pointer_cast(d_a.data());
    const cutlass::half_t* CB = thrust::raw_pointer_cast(d_b.data());
    const cutlass::half_t* CC = thrust::raw_pointer_cast(d_c.data());
    cutlass::half_t* CD = thrust::raw_pointer_cast(d_d2.data());

    const InType* A = reinterpret_cast<const InType*>(CA);
    const InType* B = reinterpret_cast<const InType*>(CB);
    const InType* C = reinterpret_cast<const InType*>(CC);
    InType* D = thrust::raw_pointer_cast(d_d.data());

    using Config = FusedTwoGemmsTraits<InType, AccType, WholeShape,
                                       CtaTileShape, WarpLayout, kSharedAccess>;

    using RegA = typename Config::RegA;
    using RegB = typename Config::RegB;
    using RegC = typename Config::RegC;
    using RegD = typename Config::RegD;
    using RegDHalf = typename Config::RegDHalf;
    using RegAcc = typename Config::RegAcc;
    using RegAccCast = typename Config::RegAccCast;

    using GIteratorA = typename Config::GIteratorA;
    using SharedA = typename Config::SharedA;
    using SharedALoader = typename Config::SharedALoader;
    using RegALoader = typename Config::RegALoader;

    using GIteratorB = typename Config::GIteratorB;
    using SharedB = typename Config::SharedB;
    using SharedBLoader = typename Config::SharedBLoader;
    using RegBLoader = typename Config::RegBLoader;

    using GIteratorC = typename Config::GIteratorC;
    using SharedC = typename Config::SharedC;
    using SharedCLoader = typename Config::SharedCLoader;
    using RegCLoader = typename Config::RegCLoader;

    using SharedD = typename Config::SharedD;
    using StoreRegD = typename Config::StoreRegD;
    using StoreSharedD = typename Config::StoreSharedD;

    using ConvertAcc = typename Config::ConvertHalf;
    using ConvertD = typename Config::ConvertD;

    int block_x = CeilDiv<kM, kTM>;
    int block_y = CeilDiv<kP, kTP>;
    int block_z = kBatch;

    dim3 grid(block_x, block_y, block_z);
    dim3 block(Config::kThreads, 1, 1);

    int shm_input = (kTM * kTK + kTK * kTN + kTN * kTP);
    int shm_output = kTM * kTP;
    int shm_size = shm_input < shm_output ? shm_output * sizeof(InType)
                                          : shm_input * sizeof(InType);

    auto ke_tilefusion =
        &ke_fused_two_gemms<InType, AccType,            //
                            GIteratorA, SharedA, RegA,  //
                            SharedALoader, RegALoader,  //
                            GIteratorB, SharedB, RegB,  //
                            SharedBLoader, RegBLoader,  //
                            GIteratorC, SharedC, RegC,  //
                            SharedCLoader, RegCLoader,  //
                            RegAcc, RegAccCast, typename Config::GlobalD,
                            SharedD, RegD, RegDHalf, StoreRegD, StoreSharedD,
                            ConvertAcc, ConvertD>;

    auto cutlass_fused_gemm =
        &cute_fused_gemm<cutlass::half_t, kWarpPerRow, kWarpPerCol, kM, kN, kK,
                         kP, kTM, kTN, kTK, kTP>;

    if (shm_size > 48 * 1024) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(ke_tilefusion),
                             hipFuncAttributeMaxDynamicSharedMemorySize,
                             shm_size);
    }

    ke_tilefusion<<<grid, block, shm_size, 0>>>(A, B, C, D, kM, kN, kK, kP, kTM,
                                                kTN, kTK, kTP);
    hipDeviceSynchronize();

    h_d = d_d;

    cutlass_fused_gemm(CA, CB, CC, CD, false, 0, 0);
    h_d2 = d_d2;

    thrust::host_vector<InType> h_acc(kM * kN * kBatch);
    thrust::fill(h_acc.begin(), h_acc.end(), 0.);
    thrust::device_vector<InType> d_acc = h_acc;

    cublas_two_gemms(kM, kN, kK, kP, kBatch, A, B, C,
                     thrust::raw_pointer_cast(d_d3.data()),
                     thrust::raw_pointer_cast(d_acc.data()), false);
    hipDeviceSynchronize();
    h_acc = d_acc;
    h_d3 = d_d3;

#ifdef DEBUG
    InType* data = thrust::raw_pointer_cast(h_d.data());
    cutlass::half_t* cutlass_data = thrust::raw_pointer_cast(h_d2.data());
    __half* cutlass_data_half = reinterpret_cast<__half*>(cutlass_data);
    __half* ground_truth = thrust::raw_pointer_cast(h_d3.data());

    const int numel = 256;
    printf("ours:\n");
    for (int i = 0; i < numel; ++i) {
        printf("%.3f, ", __half2float(data[i]));
        if (i && (i + 1) % 16 == 0) printf("\n");
    }
    printf("cutlass:\n");
    for (int i = 0; i < numel; ++i) {
        printf("%.3f, ", __half2float(cutlass_data_half[i]));
        if (i && (i + 1) % 16 == 0) printf("\n");
    }
    printf("\nground_truth:\n");
    for (int i = 0; i < numel; ++i) {
        printf("%.3f, ", __half2float(ground_truth[i]));
        if (i && (i + 1) % 16 == 0) printf("\n");
    }

    bool passed1 = check_results(data, ground_truth, kM * kP, epsilon);
    bool passed2 =
        check_results(cutlass_data_half, ground_truth, kM * kP, epsilon);
    std::cout << "passed1: " << passed1 << ", passed2: " << passed2
              << std::endl;

    if (passed1 && passed2) {
        std::cout << "[" << kM << ", " << kN << ", " << kK << ", " << kP
                  << "], batch = " << kBatch << ", passed." << std::endl;
    } else {
        std::cout << "[" << kM << ", " << kN << ", " << kK << ", " << kP
                  << "], batch = " << kBatch << ", failed." << std::endl;
    }

#endif

    CudaTimer timer;
    const int warm_up = 10;
    const int iters = 50;

    for (int i = 0; i < warm_up; ++i) {
        ke_tilefusion<<<grid, block, shm_size, 0>>>(A, B, C, D, kM, kN, kK, kP,
                                                    kTM, kTN, kTK, kTP);
    }
    hipDeviceSynchronize();

    timer.start();
    for (int i = 0; i < iters; ++i) {
        ke_tilefusion<<<grid, block, shm_size, 0>>>(A, B, C, D, kM, kN, kK, kP,
                                                    kTM, kTN, kTK, kTP);
    }
    hipDeviceSynchronize();
    float tilefusion_time = timer.stop() / iters;

    float cutlass_time =
        cutlass_fused_gemm(CA, CB, CC, CD, true, warm_up, iters);

    float cublas_time = cublas_two_gemms(
        kM, kN, kK, kP, kBatch, A, B, C, thrust::raw_pointer_cast(d_d3.data()),
        thrust::raw_pointer_cast(d_acc.data()), true);

    std::cout << "[" << kM << ", " << kN << ", " << kK << ", " << kP << "]\t["
              << kTM << ", " << kTN << ", " << kTK << ", " << kTP << "]\t"
              << cublas_time << "\t" << cutlass_time << "("
              << cutlass_time / cublas_time << ")"
              << "\t" << tilefusion_time << "(" << tilefusion_time / cublas_time
              << ")" << std::endl;
}

int main() {
    // using WarpLayout1 = tl::RowMajor<2, 1>;
    // static constexpr int kSharedAccess0 = 64;

    // std::cout << "[kM, kN, kK, kP]\t[kTM, kTN, kTK, kTP]\t[cublas "
    //              "time]\t[tilefusion time(Radio)]"
    //           << std::endl;

    // run<B2BGemmShape<256 /*M*/, 128 /*N*/, 64 /*K*/, 64 /*P*/>,
    //     B2BGemmShape<64 /*kTM*/, 64 /*kTN*/, 64 /*kTK*/, 64 /*kTP*/>,
    //     WarpLayout1, 1, kSharedAccess0>(5e-3);

    using WarpLayout2 = tl::RowMajor<4, 1>;
    static constexpr int kSharedAccess1 = 128;
    // run<B2BGemmShape<2048 /*M*/, 2048 /*N*/, 128 /*K*/, 128 /*P*/>,
    //     B2BGemmShape<64 /*kTM*/, 128 /*kTN*/, 128 /*kTK*/, 128 /*kTP*/>,
    //     WarpLayout2, 1, 64>(5e-3);

    // run<B2BGemmShape<1024 /*M*/, 1024 /*N*/, 128 /*K*/, 128 /*P*/>,
    //     B2BGemmShape<64 /*kTM*/, 128 /*kTN*/, 128 /*kTK*/, 128 /*kTP*/>,
    //     WarpLayout2, 1, 64>(5e-3);

    run<B2BGemmShape<512 /*M*/, 512 /*N*/, 128 /*K*/, 128 /*P*/>,
        B2BGemmShape<64 /*kTM*/, 128 /*kTN*/, 128 /*kTK*/, 128 /*kTP*/>,
        WarpLayout2, 1, 64>(5e-3);

    return 0;
}
